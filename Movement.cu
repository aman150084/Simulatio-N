#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <vector>

#define size 100000;                                    //space size
#define t 0.0001;

using namespace std;

float Field[size][size][size]={0};

class Body                                              //basic planet properties
{
  public:
    float mass;
    vector<float> pos (3,0);
    vector<float> vel (3,0);
};

__global__ void Movement(Body* planet,vector<float> force)
{
  
}


int main(void)
{
  int n;                                               //planet counter
  int i=0;                                             //counter variable
  int j;
  int* t_mass;
  vector<float>* t_pos (3,0);


  cout<<"Input number of bodies:\n";
  cin>>n;
  Body* List[n];
  for(i=0;i<n;i++)
  {
      List[i]=hipMalloc((void**)&Body,sizeof(Body));
      List[i]->vel = (0,0,0);
      cout<<"mass and postion of"<<i+1<<"th Body?\n";
      cin>>List[i]->mass>>List[i]->x>>List[i]->y>>List[i]->z;
      hipMemcpy(t_mass,List[i].mass,sizeof(float),hipMemcpyHostToDevice);
      for(j=0;j<3;j++)
        hipMemcpy(t_pos[j],List[i].pos[j],sizeof(float),hipMemcpyHostToDevice);
  }
}
